#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void prescan(int n, int* g_odata, int* g_idata)
        {
            int thid = threadIdx.x;
            int offset = 1;
            extern __shared__ int temp[];
            temp[2 * thid] = g_idata[2 * thid]; // load input into shared memory
             temp[2*thid+1] = g_idata[2*thid+1];
            for (int d = n >> 1; d > 0; d >>= 1)
                // build sum in place up the tree
            {
                __syncthreads();
                if (thid < d)
                {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    temp[bi] += temp[ai];
                }
                offset <<= 1;
            }
            if (thid == 0)
            {
                temp[n - 1] = 0;
            } // clear the last element
            for (int d = 1; d < n; d <<= 1) // traverse down tree & build scan
            {
                offset >>= 1;
                __syncthreads();
                if (thid < d)
                {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();
            g_odata[2 * thid] = temp[2 * thid]; // write results to device memory
            g_odata[2 * thid + 1] = temp[2 * thid + 1];
        }

        __global__ void make_exclusive(int n, int* odata, const int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) {
                return;
            }
            if (index == 0) {
                odata[index] = 0;
                return;
            }
            odata[index] = idata[index - 1];
        }

        //__global__ void pad_with_zeroes(int n, int m, int* g_data) {
        //    int index = (blockIdx.x * blockDim.x) + threadIdx.x;
        //    if (index >= n || index < m) {
        //        return;
        //    }
        //    g_data[index] = 0;
        //}

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return;
            }
            if (n == 1) {          // handle trivial case without GPU work
                odata[0] = 0;
                return;
            }
            int m = n;
            bool is_pow_two = (n & (n - 1)) == 0;
            if (!is_pow_two) {
                n = 1 << ilog2ceil(n);
            }

            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int sharedMemBytes = 2 * blockSize * sizeof(int);
            int* dev_buf_i = nullptr;
            int* dev_buf_o = nullptr;

            hipMalloc((void**)&dev_buf_i, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_buf_i failed!");
            hipMalloc((void**)&dev_buf_o, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_buf_o failed!");\

            hipMemcpy(dev_buf_i, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("MemCpy dev_buf_i failed!");

            if (!is_pow_two) {
                hipMemset(dev_buf_i + m, 0, (n - m) * sizeof(int));
            }
            timer().startGpuTimer();
            Efficient::prescan << <fullBlocksPerGrid, blockSize, sharedMemBytes >>> (n, dev_buf_o, dev_buf_i);
            checkCUDAErrorFn("prescan failed!");
            hipDeviceSynchronize();
            timer().endGpuTimer();

            hipMemcpy(odata, dev_buf_o, m * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("MemCpy dev_buf_o failed!");
            hipFree(dev_buf_o);
            checkCUDAErrorFn("CudaFree dev_buf_o failed!");
            hipFree(dev_buf_i);
            checkCUDAErrorFn("CudaFree dev_buf_i failed!");
        }

        __global__ void scatter(int n, int* bools, int* odata, const int* idata) {
            //odata already has indices
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) {
                return;
            }
            if (bools[index] == 1) {
                odata[odata[index]] = idata[index];
            }
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            //Check trivial cases:
            if (n <= 0) {
                return 0;
            }
            if (n == 1) {
                if (idata[0] != 0)
                {
                    odata[0] = idata[0];
                    return 1;
                }
                return 0;
            }
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int m = n;
            n = 1 << ilog2ceil(n);
            timer().startGpuTimer();
                // Allocate and assign input
                int* dev_buf_i = nullptr;
                hipMalloc((void**)&dev_buf_i, n * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_i failed!");
                hipMemcpy(dev_buf_i, idata, m * sizeof(int), hipMemcpyHostToDevice);
                checkCUDAErrorFn("MemCpy dev_buf_i failed!");

         
                // Allocate bools buffer
                int* dev_buf_bools = nullptr;
                hipMalloc((void**)&dev_buf_bools, n * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_bools failed!");

                // Fill bools buffer
                
                Common::kernMapToBoolean <<<fullBlocksPerGrid, blockSize >> > (m, dev_buf_bools, dev_buf_i);

                // Scan bools to output

                dim3 fullBlocksPerGrid_scan((n + blockSize - 1) / blockSize);
                int sharedMemBytes = 2 * blockSize * sizeof(int);
                int* dev_buf_indices = nullptr;

                hipMalloc((void**)&dev_buf_indices, n * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_indices failed!");

                hipMemset(dev_buf_bools + m, 0, (n - m) * sizeof(int));
                checkCUDAErrorFn("CudaMemset zeroes failed!");

                Efficient::prescan<<<fullBlocksPerGrid_scan, blockSize, sharedMemBytes>>> (n, dev_buf_indices, dev_buf_bools);
                checkCUDAErrorFn("prescan failed!");
                hipDeviceSynchronize();

                //std::swap(m, n);

                // Shorten now

                int* dev_buf_o = nullptr;
                hipMalloc((void**)&dev_buf_o, m * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_i failed!");

                Common::kernScatter << < fullBlocksPerGrid, blockSize>>> (m, dev_buf_o, dev_buf_i, dev_buf_bools, dev_buf_indices);
                checkCUDAErrorFn("efficient scatter failed!");
                hipDeviceSynchronize();

                int last_index;
                hipMemcpy(&last_index, dev_buf_indices + m - 1, sizeof(int), hipMemcpyDeviceToHost);

                int last_bool;
                hipMemcpy(&last_bool, dev_buf_bools + m - 1, sizeof(int), hipMemcpyDeviceToHost);

                int count = (last_index + last_bool);
                // Copy to CPU and Free data
                hipMemcpy(odata, dev_buf_o, count * sizeof(int), hipMemcpyDeviceToHost);
                checkCUDAErrorFn("MemCpy dev_buf_o failed!");

                hipFree(dev_buf_indices);
                checkCUDAErrorFn("CudaFree dev_buf_indices failed!");
                hipFree(dev_buf_i);
                checkCUDAErrorFn("CudaFree dev_buf_i failed!");
                hipFree(dev_buf_bools);
                checkCUDAErrorFn("CudaFree dev_buf_bools failed!");
                hipFree(dev_buf_o);
                checkCUDAErrorFn("CudaFree dev_buf_o failed!");

                timer().endGpuTimer();
                return count;
        }
    }
}
