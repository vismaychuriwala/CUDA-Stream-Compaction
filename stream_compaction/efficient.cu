#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "efficient.h"

#define NUM_BANKS 16
#define LOG_NUM_BANKS 4
#define CONFLICT_FREE_OFFSET(n)((n) >> NUM_BANKS + (n) >> (2 * LOG_NUM_BANKS))

namespace StreamCompaction {
    namespace Efficient {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }

        __global__ void prescan(int n, int* g_odata, const int* g_idata)
        {
            int thid = threadIdx.x;
            int offset = 1;
            extern __shared__ int temp[];
            int ai = thid;
            int bi = thid + (n / 2);
            int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
            int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
            temp[ai + bankOffsetA] =
                g_idata[ai];
            temp[bi + bankOffsetB] = g_idata[bi];
            for (int d = n >> 1; d > 0; d >>= 1)
                // build sum in place up the tree
            {
                __syncthreads();
                if (thid < d)
                {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    temp[bi] += temp[ai];
                }
                offset <<= 1;
            }
            if (thid == 0)
            {
                temp[n - 1 + CONFLICT_FREE_OFFSET(n - 1)] = 0;
            } // clear the last element
            for (int d = 1; d < n; d <<= 1) // traverse down tree & build scan
            {
                offset >>= 1;
                __syncthreads();
                if (thid < d)
                {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();
            g_odata[ai] = temp[ai + bankOffsetA];
            g_odata[bi] = temp[bi + bankOffsetB];
        }

        __global__ void multi_scan(int global_n, int B, int* g_odata, const int* g_idata, int* blockSums)
        {
            int thid = threadIdx.x;
            int base = B * blockIdx.x;
            int offset = 1;
            extern __shared__ int temp[];
            int ai = thid;
            int bi = thid + (B / 2);
            int ga = base + ai; // Global indexes 
            int gb = base + bi;
            int bankOffsetA = CONFLICT_FREE_OFFSET(ai);
            int bankOffsetB = CONFLICT_FREE_OFFSET(bi);
            temp[ai + bankOffsetA] =
                g_idata[ga];
            temp[bi + bankOffsetB] = g_idata[gb];
            for (int d = B >> 1; d > 0; d >>= 1)
                // build sum in place up the tree
            {
                __syncthreads();
                if (thid < d)
                {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    temp[bi] += temp[ai];
                }
                offset <<= 1;
            }
            if (thid == 0)
            {
                blockSums[blockIdx.x] =
                    temp[B - 1 + CONFLICT_FREE_OFFSET(B - 1)];
                temp[B - 1 + CONFLICT_FREE_OFFSET(B - 1)] = 0;
            } // clear the last element
            for (int d = 1; d < B; d <<= 1) // traverse down tree & build scan
            {
                offset >>= 1;
                __syncthreads();
                if (thid < d)
                {
                    int ai = offset * (2 * thid + 1) - 1;
                    int bi = offset * (2 * thid + 2) - 1;
                    ai += CONFLICT_FREE_OFFSET(ai);
                    bi += CONFLICT_FREE_OFFSET(bi);
                    int t = temp[ai];
                    temp[ai] = temp[bi];
                    temp[bi] += t;
                }
            }
            __syncthreads();
            g_odata[ga] = temp[ai + bankOffsetA];
            g_odata[gb] = temp[bi + bankOffsetB];
        }

        __global__ void make_exclusive(int n, int* odata, const int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) {
                return;
            }
            if (index == 0) {
                odata[index] = 0;
                return;
            }
            odata[index] = idata[index - 1];
        }

        __global__ void uniformAdd(int n,
            int* odata,
            const int* blockIncr,
            int B) {
            int base = blockIdx.x * B;
            int offset = blockIncr[blockIdx.x];  // scanned block sums

            int i = base + threadIdx.x;
            int j = base + threadIdx.x + (B / 2);

            if (i < n) odata[i] += offset;
            if (j < n) odata[j] += offset;
        }

        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return;
            }
            if (n == 1) {          // handle trivial case without GPU work
                odata[0] = 0;
                return;
            }
            int m = n;
            bool is_pow_two = (n & (n - 1)) == 0;
            if (!is_pow_two) {
                n = 1 << ilog2ceil(n);
            }


            int* dev_buf_i = nullptr;   //Input buffer
            int* dev_buf_o = nullptr;   //Output buffer
            int* dev_buf_blockSums = nullptr;   // Per-block Sums
            int* dev_buf_blockIncr = nullptr;   // Per-block sums scan

            // Multi Scan

            // Number of blocks and threads
            int B = 2 * blockSize;
            int numBlocks = (n + B - 1) / B;    // ceil(n / B)
            dim3 fullBlocksPerGrid(numBlocks);
            int sharedMemBytes = (B + CONFLICT_FREE_OFFSET(B)) * sizeof(int);

            // Allocate and copy Memory
            hipMalloc((void**)&dev_buf_i, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_buf_i in scan failed!");

            hipMalloc((void**)&dev_buf_o, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_buf_o in scan failed!");

            hipMemcpy(dev_buf_i, idata, m * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("MemCpy dev_buf_i failed!");


            // Inter-Block Accumulation

            // Number of blocks and threads
            int num_blocks_next_power_2 = 1 << ilog2ceil(numBlocks);
            dim3 fullBlocksPerGrid_offsets((numBlocks + blockSize - 1) / blockSize);
            int sharedMem_offsets = (num_blocks_next_power_2 + CONFLICT_FREE_OFFSET(num_blocks_next_power_2)) * sizeof(int);
            int threads_offsets = num_blocks_next_power_2 / 2;

            // Allocate and copy Memory
                hipMalloc((void**)&dev_buf_blockSums, num_blocks_next_power_2 * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_blockSums in scan failed!");

                if (num_blocks_next_power_2 > numBlocks) {
                    hipMemset(dev_buf_blockSums + numBlocks, 0,
                        (num_blocks_next_power_2 - numBlocks) * sizeof(int));
                }
                hipMalloc((void**)&dev_buf_blockIncr, num_blocks_next_power_2 * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_blockIncr in scan failed!");

            if (!is_pow_two) {
                hipMemset(dev_buf_i + m, 0, (n - m) * sizeof(int));
            }

            timer().startGpuTimer();

            // Block-Wise Multi-Scan
            Efficient::multi_scan <<<fullBlocksPerGrid, blockSize, sharedMemBytes >>> (n, B, dev_buf_o, dev_buf_i, dev_buf_blockSums);
            checkCUDAErrorFn("multi-scan failed!");
            hipDeviceSynchronize();
            if (numBlocks > 1) {
                Efficient::prescan << <fullBlocksPerGrid_offsets, threads_offsets, sharedMem_offsets >> > (num_blocks_next_power_2, dev_buf_blockIncr, dev_buf_blockSums);
                checkCUDAErrorFn("prescan of offsets failed!");
                hipDeviceSynchronize();

                uniformAdd << <numBlocks, B / 2 >> > (n, dev_buf_o, dev_buf_blockIncr, B);
                checkCUDAErrorFn("Uniform Add failed!");
                hipDeviceSynchronize();
            }

            timer().endGpuTimer();

            hipMemcpy(odata, dev_buf_o, m * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("MemCpy dev_buf_o in scan failed!");
            hipFree(dev_buf_o);
            checkCUDAErrorFn("CudaFree dev_buf_o in scan failed!");
            hipFree(dev_buf_i);
            checkCUDAErrorFn("CudaFree dev_buf_i in scan failed!");

            hipFree(dev_buf_blockSums);
            checkCUDAErrorFn("CudaFree dev_buf_blockSums in scan failed!");
            hipFree(dev_buf_blockIncr);
            checkCUDAErrorFn("CudaFree dev_buf_blockIncr in scan failed!");
        }

        /**
         * Performs stream compaction on idata, storing the result into odata.
         * All zeroes are discarded.
         *
         * @param n      The number of elements in idata.
         * @param odata  The array into which to store elements.
         * @param idata  The array of elements to compact.
         * @returns      The number of elements remaining after compaction.
         */
        int compact(int n, int* odata, const int* idata) {
            //Check trivial cases:
            if (n <= 0) {
                return 0;
            }
            if (n == 1) {
                if (idata[0] != 0)
                {
                    odata[0] = idata[0];
                    return 1;
                }
                return 0;
            }
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int m = n;
            n = 1 << ilog2ceil(n);
           
                // Allocate and assign input
                int* dev_buf_i = nullptr;
                hipMalloc((void**)&dev_buf_i, n * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_i failed!");
                hipMemcpy(dev_buf_i, idata, m * sizeof(int), hipMemcpyHostToDevice);
                checkCUDAErrorFn("MemCpy dev_buf_i failed!");

                // Allocate bools buffer
                int* dev_buf_bools = nullptr;
                hipMalloc((void**)&dev_buf_bools, n * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_bools failed!");

                //Allocate output buffer
                int* dev_buf_o = nullptr;
                hipMalloc((void**)&dev_buf_o, m * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_o failed!");

                //Allocate indices buffer
                int* dev_buf_indices = nullptr;
                hipMalloc((void**)&dev_buf_indices, n * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_indices failed!");

                hipMemset(dev_buf_bools + m, 0, (n - m) * sizeof(int));    // Padding zeroes to nearest power of two
                checkCUDAErrorFn("CudaMemset zeroes failed!");

                // Compute shared memory sizes
                //dim3 fullBlocksPerGrid_scan((n + blockSize - 1) / blockSize);
                //int sharedMemBytes = 2 * blockSize * sizeof(int);

                // Number of blocks and threads
                int B = 2 * blockSize;
                int numBlocks = (n + B - 1) / B;    // ceil(n / B)
                dim3 fullBlocksPerGrid_scan(numBlocks);
                int sharedMemBytes_scan = (B + CONFLICT_FREE_OFFSET(B)) * sizeof(int);

                // Inter-Block Accumulation

                int* dev_buf_blockSums = nullptr;   // Per-block Sums
                int* dev_buf_blockIncr = nullptr;   // Per-block sums scan

                // Number of blocks and threads
                int num_blocks_next_power_2 = 1 << ilog2ceil(numBlocks);
                dim3 fullBlocksPerGrid_offsets((numBlocks + blockSize - 1) / blockSize);
                int sharedMem_offsets = (num_blocks_next_power_2 + CONFLICT_FREE_OFFSET(num_blocks_next_power_2)) * sizeof(int);
                int threads_offsets = num_blocks_next_power_2 / 2;

                // Allocate and copy Memory
                hipMalloc((void**)&dev_buf_blockSums, num_blocks_next_power_2 * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_blockSums in scan failed!");

                if (num_blocks_next_power_2 > numBlocks) {
                    hipMemset(dev_buf_blockSums + numBlocks, 0,
                        (num_blocks_next_power_2 - numBlocks) * sizeof(int));
                }
                hipMalloc((void**)&dev_buf_blockIncr, num_blocks_next_power_2 * sizeof(int));
                checkCUDAErrorFn("hipMalloc dev_buf_blockIncr in scan failed!");

                
                timer().startGpuTimer();
                //------------------------------------GPU--------------------------------------
                // Fill bools buffer
                Common::kernMapToBoolean <<<fullBlocksPerGrid, blockSize >> > (m, dev_buf_bools, dev_buf_i);
                hipDeviceSynchronize();

                // Scan bools to output
                Efficient::multi_scan << <fullBlocksPerGrid_scan, blockSize, sharedMemBytes_scan >> > (n, B, dev_buf_indices, dev_buf_bools, dev_buf_blockSums);
                checkCUDAErrorFn("multi-scan failed!");
                hipDeviceSynchronize();
                if (numBlocks > 1) {
                    Efficient::prescan << <fullBlocksPerGrid_offsets, threads_offsets, sharedMem_offsets >> > (num_blocks_next_power_2, dev_buf_blockIncr, dev_buf_blockSums);
                    checkCUDAErrorFn("prescan of offsets failed!");
                    hipDeviceSynchronize();

                    uniformAdd << <numBlocks, B / 2 >> > (n, dev_buf_indices, dev_buf_blockIncr, B);
                    checkCUDAErrorFn("Uniform Add failed!");
                    hipDeviceSynchronize();
                }

                // Compact now
                Common::kernScatter << < fullBlocksPerGrid, blockSize>>> (m, dev_buf_o, dev_buf_i, dev_buf_bools, dev_buf_indices);
                checkCUDAErrorFn("efficient scatter failed!");
                hipDeviceSynchronize();
                //----------------------------------------------------------------------------
                timer().endGpuTimer();

                // Compute size of compacted array
                int last_index;
                hipMemcpy(&last_index, dev_buf_indices + m - 1, sizeof(int), hipMemcpyDeviceToHost);

                int last_bool;
                hipMemcpy(&last_bool, dev_buf_bools + m - 1, sizeof(int), hipMemcpyDeviceToHost);

                long long int count = (last_index + last_bool);

                // Copy output to CPU
                hipMemcpy(odata, dev_buf_o, count * sizeof(int), hipMemcpyDeviceToHost);
                checkCUDAErrorFn("MemCpy dev_buf_o failed! (Copying output to cpu)");

                // Free data
                hipFree(dev_buf_indices);
                checkCUDAErrorFn("CudaFree dev_buf_indices failed!");
                hipFree(dev_buf_i);
                checkCUDAErrorFn("CudaFree dev_buf_i failed!");
                hipFree(dev_buf_bools);
                checkCUDAErrorFn("CudaFree dev_buf_bools failed!");
                hipFree(dev_buf_o);
                checkCUDAErrorFn("CudaFree dev_buf_o failed!");

                hipFree(dev_buf_blockSums);
                checkCUDAErrorFn("CudaFree dev_buf_blockSums in scan failed!");
                hipFree(dev_buf_blockIncr);
                checkCUDAErrorFn("CudaFree dev_buf_blockIncr in scan failed!");
            //timer().startGpuTimer();
            //int count = -1;
            //timer().endGpuTimer();
                return count;
        }
    }
}
