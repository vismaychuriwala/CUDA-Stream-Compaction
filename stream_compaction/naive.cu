#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "common.h"
#include "naive.h"

namespace StreamCompaction {
    namespace Naive {
        using StreamCompaction::Common::PerformanceTimer;
        PerformanceTimer& timer()
        {
            static PerformanceTimer timer;
            return timer;
        }
        __global__ void onestep(int n, int* odata, const int* idata,int d) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) {
                return;
            }
            int two_pow_d_minus_one = 1 << (d - 1);
            if (index >= two_pow_d_minus_one) {
                odata[index] = idata[index - two_pow_d_minus_one] + idata[index];
            }
            else {
                odata[index] = idata[index];
            }
        }

        __global__ void make_exclusive(int n, int* odata, const int* idata) {
            int index = (blockIdx.x * blockDim.x) + threadIdx.x;
            if (index >= n) {
                return;
            }
            if (index == 0) {
                odata[index] = 0;
                return;
            }
            odata[index] = idata[index - 1];
        }
        /**
         * Performs prefix-sum (aka scan) on idata, storing the result into odata.
         */
        void scan(int n, int *odata, const int *idata) {
            if (n <= 0) {
                return;
            }
            if (n == 1) {          // handle trivial case without GPU work
                odata[0] = 0;
                return;
            }
            dim3 fullBlocksPerGrid((n + blockSize - 1) / blockSize);
            int* dev_bufA = nullptr;
            int* dev_bufB = nullptr;

            hipMalloc((void**)&dev_bufA, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_buf_A failed!");

            hipMalloc((void**)&dev_bufB, n * sizeof(int));
            checkCUDAErrorFn("hipMalloc dev_buf_B failed!");

            hipMemcpy(dev_bufA, idata, n * sizeof(int), hipMemcpyHostToDevice);
            checkCUDAErrorFn("MemCpy dev_buf_A failed!");

            timer().startGpuTimer();
            int num_iter = ilog2ceil(n);

            for (int d = 1; d <= num_iter; d++) {
                onestep << <fullBlocksPerGrid, blockSize >> > (n, dev_bufB, dev_bufA, d);
                checkCUDAErrorFn("onestep Naive failed!");
                hipDeviceSynchronize();
                std::swap(dev_bufA, dev_bufB);  // Output in dev_buf_A
            }
            // Inclusive Scan to Exclusive
            make_exclusive<<<fullBlocksPerGrid, blockSize >> > (n, dev_bufB, dev_bufA);  // Exclusive scan in dev_buf_B
            checkCUDAErrorFn("Shift_right failed!");
            hipDeviceSynchronize();
            timer().endGpuTimer();

            hipMemcpy(odata, dev_bufB, n * sizeof(int), hipMemcpyDeviceToHost);
            checkCUDAErrorFn("MemCpy dev_buf_B failed!");
            hipFree(dev_bufA);
            checkCUDAErrorFn("CudaFree dev_buf_A failed!");
            hipFree(dev_bufB);
            checkCUDAErrorFn("CudaFree dev_buf_B failed!");
        }
    }
}
